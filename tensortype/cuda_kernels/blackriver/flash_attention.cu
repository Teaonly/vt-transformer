#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_cooperative_groups.h>
#include <cstddef>
#include <cstdio>

// just declare
namespace vt { namespace cuda {
template <typename T>
int flash_attention(const T* query, T* key, T* value, int batch, int heads, int length, int hidden, hipStream_t stream);
}}

namespace vt { namespace cuda {
    
}}

